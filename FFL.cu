#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int N = 4;
__global__ void forward_propagation_fc(float* input, float* weights, float* bias, float* output) {
         int i = threadIdx.x;
         float sum = 0.0f;
         for(int j = 0; j < N; j++){
         sum += bias[j] + weights[i*N + j] * input[j];
        }
        output[i] = sum;
}

int main()
{
     // Allocate memory for arrays
    float* input = (float*)malloc(N * sizeof(float));
    float* weights = (float*)malloc(N*N * sizeof(float));
    float* output = (float*)malloc(N * sizeof(float));
    float* biases = (float*)malloc(N * sizeof(float));
    // Initialize all arrays
    for (int i = 0; i < N; i++) {
        input[i] = i;
        for(int j = 0; j < N; j++){
        weights[i*N + j] = 0.5f;}
        output[i] = 0.0f;
        biases[i] = 0.0f;
    }
    // Allocate CUDA Memory
    float* d_input;
    hipMalloc(&d_input, N * sizeof(float));
    float* d_weights;
    hipMalloc(&d_weights, N*N * sizeof(float));
    float* d_output;
    hipMalloc(&d_output, N * sizeof(float));
    float* d_biases;
    hipMalloc(&d_biases, N * sizeof(float));
    // Copy the required parameters to device
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, N*N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_biases, biases, N * sizeof(float), hipMemcpyHostToDevice);
    // Launch the kernel.
    dim3 blocks(1);
    dim3 threads(N);
    forward_propagation_fc<<<blocks, threads>>>(d_input, d_weights, d_biases, d_output);
    // Copy the output array back to the host.
    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);
    // Print input array
    for (int i = 0; i < N; i++) {
        printf("%f ", input[i]);
    }
    printf("\n\n\n");
    // Print weights
    for (int i = 0; i < N; i++) {
        for (int j = 0;j < N; j++){
        printf("%f ", weights[i*N + j]);
    }}
    printf("\n\n\n");
    // Print the output array.
    for (int i = 0; i < N; i++) {
        printf("%f ", output[i]);
    }

    // Free CUDA memory.
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_output);
    hipFree(d_biases);
    // Free host memory.
    free(input);
    free(weights);
    free(output);
    free(biases);
    return 0;
}