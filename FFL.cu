#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void forward_propagation_fc(float* input, float* weights, float* bias, float* output) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  output[i] = bias[i] + weights[i] * input[i];
}


void initialize(float *matrix, int matrix_M, int matrix_N){
    for (int i = 0; i < matrix_M; i++){
        for (int j = 0; j < matrix_N; j++){
            matrix[(i*matrix_N) + j] = j + i;
        }
    }
}

void check_matrix(float *matrix, int matrix_M, int matrix_N){
    for (int i=0; i<matrix_M; i++){
        for (int j=0; j<matrix_N; j++)
        {
                printf("%.2f", matrix[(i*matrix_M)+j]);
                printf(" ");
        }
        printf("\n");
    }
    printf("\n");
}


int main(){

     // Allocate the input and output arrays.
    float* input = (float*)malloc(N * sizeof(float));
    float* weights = (float*)malloc(N*N * sizeof(float));
    float* output = (float*)malloc(N * sizeof(float));
    float* biases = (float*)malloc(N * sizeof(float));
    // Initialize the input and output arrays.
    for (int i = 0; i < N; i++) {
        input[i] = i;
        for(int j = 0; j < N; j++){
        weights[i*N + j] = 0.5f;}
        output[i] = 0.0f;
        biases[i] = 0.0f;
    }

    // Allocate the CUDA memory for the input and output arrays.
    float* d_input;
    hipMalloc(&d_input, N * sizeof(float));
    float* d_weights;
    hipMalloc(&d_weights, N*N * sizeof(float));
    float* d_output;
    hipMalloc(&d_output, N * sizeof(float));
    float* d_biases;
    hipMalloc(&d_biases, N * sizeof(float));
    // Copy the input and output arrays to the CUDA device.
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, N*N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_biases, biases, N * sizeof(float), hipMemcpyHostToDevice);
    // Launch the kernel.
    dim3 gridsize(1);
    dim3 blocksize(N);
    //fully_connected_forward<<<blocks, threads>>>(d_input, d_weights, d_output, 1, N, N);
    forward_propagation_fc<<<gridsize, blocksize>>>(d_input, d_weights, d_biases, d_output);
 // Copy the output array back to the host.
    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%f ", input[i]);
    }
    printf("\n\n\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0;j < N; j++){
        printf("%f ", weights[i*N + j]);
    }}
    printf("\n\n\n");
    // Print the output array.
    for (int i = 0; i < N; i++) {
        printf("%f ", output[i]);
    }

    // Free the CUDA memory.
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_output);
    hipFree(d_biases);
    // Free the host memory.
    free(input);
    free(weights);
    free(output);
    free(biases);

    return 0;



}
