#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

const int N = 4;
const int bs = 1;
const int lambda = 14;

__global__ void backward_propagation_fc_lastlayer(float* sigmoid_output,int* labels,float* delta)
{
int i = threadIdx.x;
delta[i] = (sigmoid_output[i] - labels[i])/bs;
}

__global__ void backward_propagation_fc(float* sigmoid_output,float* delta_next,float* delta_curr,float* weights)
{
 int i = blockIdx.x * blockDim.x  + threadIdx.x;
 int j = blockIdx.x;
 delta_curr[i] += sigmoid_output[j]*delta_next[i % blockDim.x]; 
 delta_curr[i] /= bs;
 //delta_curr[(i % blockDim.x) + j*blockDim.x] += lambda*weights[(i % blockDim.x) + j*blockDim.x];
 delta_curr[i] += lambda*weights[i*N + j];
}


int main(){
float* sigmoid_output = (float*)malloc(N * sizeof(float));
float* delta_curr = (float*)malloc(N*N * sizeof(float));
float* delta_next = (float*)malloc(N * sizeof(float));
float* weights = (float*)malloc(N*N * sizeof(float));
//float* lambda = (float*)malloc(1 * sizeof(float));

//lambda[0] = 0.1;
for (int i = 0; i < N; i++){
    sigmoid_output[i] = i;
    cout << i << endl;
    delta_next[i] = N + i;
    for (int j = 0; j < N; j++){ 
        delta_curr[i*N + j] = 0.0f;
        weights[i*N + j] = 1.0f;
    }
}
float* d_sig_op;
hipMalloc(&d_sig_op, N * sizeof(float));
float* d_delta_curr;
hipMalloc(&d_delta_curr, N*N * sizeof(float));
float* d_delta_next;
hipMalloc(&d_delta_next, N * sizeof(float));
float* d_weights;
hipMalloc(&d_weights, N*N * sizeof(float));


hipMemcpy(d_sig_op, sigmoid_output, N * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_delta_curr, delta_curr, N*N * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_delta_next, delta_next, N * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_weights, weights, N*N * sizeof(float), hipMemcpyHostToDevice);

dim3 blocks(N);
dim3 threads(N);
backward_propagation_fc<<<blocks, threads>>>(d_sig_op, d_delta_next, d_delta_curr,d_weights);
hipMemcpy(delta_curr, d_delta_curr, N * N *sizeof(float), hipMemcpyDeviceToHost);

// Print input array
    for (int i = 0; i < N; i++) {
        printf("%f ", sigmoid_output[i]);
    }
printf("\n\n\n");
// Delta_Curr_next
for (int i = 0; i < N; i++) {
    printf("%f ", delta_next[i]);
}
printf("\n\n");
// Print weights
for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++){
    printf("%f ", weights[i*N + j]);
}}
printf("\n\n");
// Print output
for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++){
    printf("%f ", delta_curr[i*N + j]);
}}
// Free CUDA memory.
hipFree(d_sig_op);
hipFree(d_delta_next);
hipFree(d_delta_curr);
hipFree(d_weights);
// Free host memory.
free(sigmoid_output);
free(delta_curr);
free(delta_next);
free(weights);
return 0;
}