#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <vector>
#include <memory>


class FeedForwardLayer
{
public:

float *weights, *bias, *Op, lr, *dev_op,*dev_wt;
int ht, wdth, op_sz;

FeedForwardLayer::FeedForwardLayer(){
    wdth = 28;
    ht = 28;
    int op_sz = 28*28;

    float h_biases[28];
    float h_weights[28][28];
    
    hipMalloc(&Op, sizeof(float)*op_sz);
    hipMalloc(&weights, sizeof(float)*wdth*ht);
    hipMalloc(&bias, sizeof(float)*ht);
    hipMalloc(&dev_op, sizeof(float) * op_sz);
    hipMalloc(&dev_wt, sizeof(float) * wdth*ht);

    for (int i = 0; i < ht; ++i) {
		h_biases[i] = 0.5f - float(rand()) / float(RAND_MAX);
		/*h_bias[i] = 0.0f;*/

		for (int j = 0; j < wdth; ++j) {
			h_weights[i][j] = 0.5f - float(rand()) / float(RAND_MAX);
			/*h_weight[i][j] = 0.05f;*/
		}
	}
    hipMemcpy(bias, h_biases, sizeof(float) * ht, hipMemcpyHostToDevice);
	hipMemcpy(weights, h_weights, sizeof(float) * wdth * ht, hipMemcpyHostToDevice);
}
__global__ void forward(float* O, float* X, float *W, float *b, int W_x, int W_y, int X_x, int X_y){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int O_x = X_x; // Is it transposed?
    int O_y = W_y; // Is it transposed?

    //O[i + j*O_x] = W[i + j*O_x]*X[i + j*O_x] + b[i + j*O_x];
    float O_val = 0;
    if (row < O_y && col < O_x){
        for (int i = 0; i < W_x; i++){
            O_val += W[row * W_x + i] * X[i * X_x + col];
        }
        O[row*O_x + col] = O_val + b[row];
    }
    Op = O;
}
};
