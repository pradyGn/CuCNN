#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Function to read the MNIST dataset.
void read_mnist_dataset(float* train_images, float* train_labels, float* test_images, float* test_labels) {

  // Open the MNIST files.
  FILE* train_images_file = fopen("./MNIST/MNIST_for_C/data/train-images.idx3-ubyte", "rb");
  FILE* train_labels_file = fopen("./MNIST/MNIST_for_C/data/train-labels.idx1-ubyte", "rb");
  FILE* test_images_file = fopen("./MNIST/MNIST_for_C/data/t10k-images.idx3-ubyte", "rb");
  FILE* test_labels_file = fopen("./MNIST/MNIST_for_C/data/t10k-labels.idx1-ubyte", "rb");

  // Check if the files were opened successfully.
  if (train_images_file == NULL || train_labels_file == NULL || test_images_file == NULL || test_labels_file == NULL) {
    printf("Could not open MNIST files.\n");
    exit(1);
  }

  // Read the number of training images.
  int num_train_images;
  fread(&num_train_images, sizeof(int), 1, train_images_file);

  // Read the number of test images.
  int num_test_images;
  fread(&num_test_images, sizeof(int), 1, test_images_file);

  // Read the training images.
  for (int i = 0; i < num_train_images; i++) {
    unsigned char image[784];
    fread(image, sizeof(unsigned char), 784, train_images_file);
    for (int j = 0; j < 784; j++) {
      train_images[i * 784 + j] = image[j] / 255.0f;
    }
  }

  // Read the training labels.
  for (int i = 0; i < num_train_images; i++) {
    unsigned char label;
    fread(&label, sizeof(unsigned char), 1, train_labels_file);
    train_labels[i] = label;
  }

  // Read the test images.
  for (int i = 0; i < num_test_images; i++) {
    unsigned char image[784];
    fread(image, sizeof(unsigned char), 784, test_images_file);
    for (int j = 0; j < 784; j++) {
      test_images[i * 784 + j] = image[j] / 255.0f;
    }
  }

  // Read the test labels.
  for (int i = 0; i < num_test_images; i++) {
    unsigned char label;
    fread(&label, sizeof(unsigned char), 1, test_labels_file);
    test_labels[i] = label;
  }

  // Close the files.
  fclose(train_images_file);
  fclose(train_labels_file);
  fclose(test_images_file);
  fclose(test_labels_file);
}

int main() {

  // Initialize the arrays.
  float* train_images = (float*)malloc(sizeof(float) * 60000 * 784);
  float* train_labels = (float*)malloc(sizeof(float) * 60000);
  float* test_images = (float*)malloc(sizeof(float) * 10000 * 784);
  float* test_labels = (float*)malloc(sizeof(float) * 10000);

  // Read the MNIST dataset.
  read_mnist_dataset(train_images, train_labels, test_images, test_labels);

  // Print out the first image.
  for (int i = 0; i < 784; i++) {
    printf("%f ", train_images[i]);
  }
  printf("\n");

  // Print out the first label.
  printf("%f\n", train_labels[0]);

  // Free the memory.
  free(train_images);
  free(train_labels);
  free(test_images);
  free(test_labels);

  return 0;
}