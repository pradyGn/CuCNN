#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <cmath>


__global__ void sigmoid_function(float* input, float* output){
    int i = threadIdx.x;

    output[i] = 1/(1 + exp(-1*input[i]));
}