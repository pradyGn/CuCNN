#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>


#define input_M 3
#define input_N 3
#define filter_M 2
#define filter_N 2
#define output_M 4
#define output_N 4

__global__ void convolutional_layer2D (float *filter, float *input, float *output)
{
    int i = threadIdx.x;
    int j = blockIdx.x;

    int input_pos = i + (j*input_N);
    //int output_pos = (i + (filter_N - 1)) + ((j + (filter_M - 1)) * output_N);
    
    for (int m = 0; m < filter_M; m++){
        for (int n = 0; n < filter_N; n++){
            //int output_pos = (i + (filter_N - 1) - n) + (j + (filter_M - 1) - m) * output_N;
            int output_pos = (i + (filter_N - 1) - n) + ((j + (filter_M - 1)) * output_N);
            int filter_pos = (m*filter_N) + n;
            output[output_pos] += filter[filter_pos] * input[input_pos];
        }
    }
    
    
    


}

void initialize(float *matrix, int matrix_M, int matrix_N){
    for (int i = 0; i < matrix_M; i++){
        for (int j = 0; j < matrix_N; j++){
            matrix[(i*matrix_N) + j] = j + i;
        }
    }
}


int main(){

    float *d_output, *h_output, *d_filter, *h_filter, *d_input, *h_input;

    h_output = (float*)malloc(sizeof(float) * (output_M * output_M));
    h_filter = (float*)malloc(sizeof(float) * (filter_M * filter_M));
    h_input = (float*)malloc(sizeof(float) * (input_M * input_M));

    initialize(h_filter, filter_M, filter_M);
    initialize(h_input, input_M, input_M);
    
    for (int i = 0; i < output_N; i++){
        for (int j = 0; j < output_N; j++){
            h_output[(i*output_N) + j] = 0;
        }
    }

    for (int i=0; i<filter_M; i++){
        for (int j=0; j<filter_M; j++)
        {
                printf("%.2f", h_filter[(i*filter_M)+j]);
                printf(" ");
        }
        printf("\n");
    }

    printf("\n");

    for (int i=0; i<input_M; i++){
        for (int j=0; j<input_M; j++)
        {
                printf("%.2f", h_input[(i*input_M)+j]);
                printf(" ");
        }
        printf("\n");
    }

    printf("\n");

    hipMalloc((void**)&d_output, sizeof(float) * (output_M * output_M));
    hipMalloc((void**)&d_filter, sizeof(float) * (filter_M * filter_M));
    hipMalloc((void**)&d_input, sizeof(float) * (input_M * input_M));

    hipMemcpy(d_filter, h_filter, sizeof(float) * (filter_M * filter_M), hipMemcpyHostToDevice);
    hipMemcpy(d_input, h_input, sizeof(float) * (input_M * input_M), hipMemcpyHostToDevice);
    hipMemcpy(d_output, h_output, sizeof(float) * (output_M * output_M), hipMemcpyHostToDevice);

    
    dim3 gridsize(input_M);
    dim3 blocksize(input_M);

    convolutional_layer2D <<<gridsize, blocksize>>>(d_filter, d_input, d_output);

    hipMemcpy(h_output, d_output, sizeof(float) * (output_M * output_M), hipMemcpyDeviceToHost);

    for (int i=0; i<output_M; i++){
        for (int j=0; j<output_M; j++)
        {
                printf("%.2f", h_output[(i*output_M)+j]);
                printf(" ");
        }
        printf("\n");
    }
    

    hipFree(d_output);
    hipFree(d_filter);
    hipFree(d_input);

    free(h_output);
    free(h_filter);
    free(h_input);

    

}