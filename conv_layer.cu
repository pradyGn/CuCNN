#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define input_M 8
#define input_N 8
#define filter_M 3
#define filter_N 3
#define output_M 6
#define output_N 6

__global__ void convolutional_layer2D (float *filter, float *input, float *output, float bias)
{
    int i = threadIdx.x;
    int j = blockIdx.x;

    int input_pos = i + (j*input_N);

    float sum = 0;
    //int output_pos = (i + (filter_N - 1) - n) + (j + (filter_M - 1) - m) * output_N;
    //int filter_pos = (m*filter_N) + n;
    
    for (int m = 0; m < filter_M; m++){
        for (int n = 0; n < filter_N; n++){

            sum += filter[m * filter_N + n] * input[input_pos + n + m*(input_N)];

        }
    }
    
    int output_pos = i + (j*output_N);

    output[output_pos] = sum + bias;
    
    


}

void initialize(float *matrix, int matrix_M, int matrix_N){
    for (int i = 0; i < matrix_M; i++){
        for (int j = 0; j < matrix_N; j++){
            matrix[(i*matrix_N) + j] = j + i;
        }
    }
}

void check_matrix(float *matrix, int matrix_M, int matrix_N){
    for (int i=0; i<matrix_M; i++){
        for (int j=0; j<matrix_N; j++)
        {
                printf("%.2f", matrix[(i*matrix_M)+j]);
                printf(" ");
        }
        printf("\n");
    }
    printf("\n");
}


int main(){

    float *d_output, *h_output, *d_filter, *h_filter, *d_input, *h_input;
    float *h_bias, *d_bias;

    h_output = (float*)malloc(sizeof(float) * (output_M * output_M));
    h_filter = (float*)malloc(sizeof(float) * (filter_M * filter_M));
    h_input = (float*)malloc(sizeof(float) * (input_M * input_M));
    h_bais = (float*)malloc(sizeof(float));

    h_bias = 0.1;
    initialize(h_filter, filter_M, filter_M);
    initialize(h_input, input_M, input_M);
    
    for (int i = 0; i < output_N; i++){
        for (int j = 0; j < output_N; j++){
            h_output[(i*output_N) + j] = 0;
        }
    }

    check_matrix(h_filter, filter_M, filter_M);
    check_matrix(h_input, input_M, input_M);


    hipMalloc((void**)&d_output, sizeof(float) * (output_M * output_M));
    hipMalloc((void**)&d_filter, sizeof(float) * (filter_M * filter_M));
    hipMalloc((void**)&d_input, sizeof(float) * (input_M * input_M));
    hipMalloc((void**)&d_bias, sizeof(float));

    hipMemcpy(d_filter, h_filter, sizeof(float) * (filter_M * filter_M), hipMemcpyHostToDevice);
    hipMemcpy(d_input, h_input, sizeof(float) * (input_M * input_M), hipMemcpyHostToDevice);
    hipMemcpy(d_output, h_output, sizeof(float) * (output_M * output_M), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, h_bias, sizeof(float), hipMemcpyHostToDevice);

    
    dim3 gridsize(output_M);
    dim3 blocksize(output_M);

    convolutional_layer2D <<<gridsize, blocksize>>>(d_filter, d_input, d_output, d_bias);

    hipMemcpy(h_output, d_output, sizeof(float) * (output_M * output_M), hipMemcpyDeviceToHost);

    for (int i=0; i<output_M; i++){
        for (int j=0; j<output_M; j++)
        {
                printf("%.2f", h_output[(i*output_M)+j]);
                printf(" ");
        }
        printf("\n");
    }
    

    hipFree(d_output);
    hipFree(d_filter);
    hipFree(d_input);

    free(h_output);
    free(h_filter);
    free(h_input);

    

}