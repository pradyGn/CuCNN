#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

void add_padding(float *input, float *padded_input, int padding_dim, int input_N, int padded_input_N){

    for (int i = 0; i<padded_input_N; i++){
        for (int j = 0; j<padded_input_N; j++){
            padded_input[i*padded_input_N + j] = 0;
        }
    }

    for (int i = padding_dim; i<(input_N + padding_dim); i++){
        cout << i << endl;
        for (int j = padding_dim; j<(input_N + padding_dim); j++){
            padded_input[i*padded_input_N + j] = input[(i-padding_dim)*input_N + (j-padding_dim)];
        }
    }


}

void initialize(float *matrix, int matrix_M, int matrix_N){
    for (int i = 0; i < matrix_M; i++){
        for (int j = 0; j < matrix_N; j++){
            matrix[(i*matrix_N) + j] = j + i + 1;
        }
    }
}


void check_matrix(float *matrix, int matrix_M, int matrix_N){
    for (int i=0; i<matrix_M; i++){
        for (int j=0; j<matrix_N; j++)
        {
                printf("%.2f", matrix[(i*matrix_M)+j]);
                printf(" ");
        }
        printf("\n");
    }
    printf("\n");
}


int main(){
    float *input, *padded_input;

    int padding_dim = 2;
    int input_N = 2;
    int padded_input_N = padding_dim + input_N;

    padded_input = (float*)malloc(sizeof(float) * (padded_input_N * padded_input_N));
    input = (float*)malloc(sizeof(float) * (input_N * input_N));

    initialize(input, input_N, input_N);
    check_matrix(input, input_N, input_N);

    add_padding(input, padded_input, padding_dim, input_N, padded_input_N);
    check_matrix(padded_input, padded_input_N, padded_input_N);


}