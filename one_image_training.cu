#include <Mnist_test.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main(){

    float* train_images = (float*)malloc(sizeof(float) * 60000 * 784);
    float* train_labels = (float*)malloc(sizeof(float) * 60000);
    float* test_images = (float*)malloc(sizeof(float) * 10000 * 784);
    float* test_labels = (float*)malloc(sizeof(float) * 10000);

    get_image_data(train_images, train_labels, test_images, test_labels)

    //Print out the first image.
    for (int i = 0; i < 784; i++) {
      printf("%f ", train_images[i]);
    }
    printf("\n");

}