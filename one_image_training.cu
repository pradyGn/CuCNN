#include "Mnist_test.h"
#include "convolution.h"
#include "constants.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main(){

    float* h_train_images = (float*)malloc(sizeof(float) * 60000 * 784);
    float* h_train_labels = (float*)malloc(sizeof(float) * 60000);
    float* h_test_images = (float*)malloc(sizeof(float) * 10000 * 784);
    float* h_test_labels = (float*)malloc(sizeof(float) * 10000);

    float *d_output, *h_output, *d_filter, *h_filter, *h_bias, *d_bias;


    get_image_data(h_train_images, h_train_labels, h_test_images, h_test_labels);

    h_output = (float*)malloc(sizeof(float) * 60000 * (output_N * output_N));
    


    // bias initization and allocation
    h_bias = (float*)malloc(sizeof(float) * (filter_M * filter_M));
    initialize_filter(h_bias, filter_M, filter_M);
    hipMalloc((void**)&d_bias, sizeof(float) * (filter_M * filter_M));
    hipMemcpy(d_bias, h_bias, sizeof(float) * (filter_M * filter_M), hipMemcpyHostToDevice);
    check_matrix(h_bias, filter_M, filter_M);




    // filter initization and allocation
    h_filter = (float*)malloc(sizeof(float) * (filter_M * filter_M));
    initialize_filter(h_filter, filter_M, filter_M);
    hipMalloc((void**)&d_filter, sizeof(float) * (filter_M * filter_M));
    hipMemcpy(d_filter, h_filter, sizeof(float) * (filter_M * filter_M), hipMemcpyHostToDevice);
    check_matrix(h_filter, filter_M, filter_M);


    for (int i = 0; i < 2; i++){

        initialize_output(h_output[784*i], output_N, output_N);


        hipMalloc((void**)&d_output, sizeof(float) * (output_N * output_N));
        hipMalloc((void**)&d_train_image, sizeof(float) * 784);


        hipMemcpy(d_output, h_output[784*i], sizeof(float) * (output_N * output_N), hipMemcpyHostToDevice);
        hipMemcpy(d_train_image, h_train_images[784*i], sizeof(float) * 784, hipMemcpyHostToDevice);
        

        dim3 gridsize(output_M);
        dim3 blocksize(output_M);

        convolutional_layer2D <<<gridsize, blocksize>>>(d_filter, d_train_image, d_output, d_bias);

        hipMemcpy(h_output[784*i], d_output, sizeof(float) * (output_M * output_M), hipMemcpyDeviceToHost);

        if (i == 1){
            check_matrix(h_train_images[784*i], input_M, input_M);
            check_matrix(h_output[784*i], output_M, output_M);
        }


        hipFree(d_output);
        hipFree(d_train_image);

    }
    
    hipFree(d_filter);
    hipFree(d_bias);



    // Free the memory.
    free(h_train_images);
    free(h_train_labels);
    free(h_test_images);
    free(h_test_labels);
    free(h_output);
    free(h_bias);
    free(h_filter);

    return 0;

}