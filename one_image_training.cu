#include "Mnist_test.h"
#include "convolution.h"
#include "constants.h"
#include "dense.h"
#include "activation_fn.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

int main(){

    float* h_train_images = (float*)malloc(sizeof(float) * 60000 * 784);
    int* h_train_labels = (int*)malloc(sizeof(int) * 60000);
    float* h_test_images = (float*)malloc(sizeof(float) * 10000 * 784);
    int* h_test_labels = (int*)malloc(sizeof(int) * 10000);

    float  *h_output, *d_filter, *h_filter, *h_bias_conv, *d_bias_conv, *d_bias_dense, *h_bias_dense, *h_weights, *d_weights;


    float *h_dense_output;


    get_image_data(h_train_images, h_train_labels, h_test_images, h_test_labels);

    h_output = (float*)malloc(sizeof(float) * 60000 * (output_N * output_N));
    h_dense_output = (float*)malloc(sizeof(float) * 60000 * (dense_output_M * 1));

    // bias initization and allocation
    h_bias_conv = (float*)malloc(sizeof(float) * (filter_M * filter_M));
    initialize_filter(h_bias_conv, filter_M, filter_M);
    hipMalloc((void**)&d_bias_conv, sizeof(float) * (filter_M * filter_M));
    hipMemcpy(d_bias_conv, h_bias_conv, sizeof(float) * (filter_M * filter_M), hipMemcpyHostToDevice);
    //check_matrix(h_bias_conv, filter_M, filter_M);




    // filter initization and allocation
    h_filter = (float*)malloc(sizeof(float) * (filter_M * filter_M));
    initialize_filter(h_filter, filter_M, filter_M);
    hipMalloc((void**)&d_filter, sizeof(float) * (filter_M * filter_M));
    hipMemcpy(d_filter, h_filter, sizeof(float) * (filter_M * filter_M), hipMemcpyHostToDevice);
    //check_matrix(h_filter, filter_M, filter_M);

    // Initialize and allocate weights and bias for Dense layer
    h_weights = (float*)malloc(sizeof(float) * (dense_output_M * (output_M * output_M)));
    h_bias_dense = (float*)malloc(sizeof(float) * (dense_output_M));
    initialize_dense_weights_and_bias(h_weights, h_bias_dense);
    hipMalloc((void**)&d_weights, sizeof(float) * (dense_output_M * (output_M * output_M)));
    hipMalloc((void**)&d_bias_dense, sizeof(float) * dense_output_M);
    hipMemcpy(d_weights, h_weights, sizeof(float) * (dense_output_M * (output_M * output_M)), hipMemcpyHostToDevice);
    hipMemcpy(d_bias_dense, h_bias_dense, sizeof(float) * dense_output_M, hipMemcpyHostToDevice);
    //check_matrix(h_bias_dense, 1, dense_output_M);
    //check_matrix(h_weights, dense_output_M, (output_M * output_M));

    for (int i = 0; i < 1; i++){

        initialize_output(&h_output[784*i], output_N, output_N);
        initialize_dense_output(&h_dense_output[10*i]);
        



        int *d_train_label;
        float *d_train_image, *h_delta_ll, *d_delta_ll, *d_delta_curr, *h_delta_curr;
        float *d_dense_output, *d_output, *d_dense_grad_input;




        h_delta_ll = (float*)malloc(sizeof(float) * dense_output_M*1);
        h_delta_curr = (float*)malloc(sizeof(float) * dense_output_M*(output_M*output_M));
        initialize_dense_output(h_delta_ll);




        hipMalloc((void**)&d_delta_curr, sizeof(float) * dense_output_M*(output_M*output_M));
        hipMalloc((void**)&d_delta_ll, sizeof(float) * dense_output_M*1);
        hipMalloc((void**)&d_output, sizeof(float) * (output_N * output_N));
        hipMalloc((void**)&d_train_image, sizeof(float) * 784);
        hipMalloc((void**)&d_dense_output, sizeof(float) * dense_output_M);
        hipMalloc((void**)&d_train_label, sizeof(int) * dense_output_M);
        hipMalloc((void**)&d_dense_grad_input, sizeof(float) * dense_output_M);

        // One hot labels
        int* one_hot_label = (int*)malloc(sizeof(int) * dense_output_M);
        for (int j = 0; j < dense_output_M; j++) {
            one_hot_label[j] = 0;
        }
        one_hot_label[h_train_labels[i]] = 1;

        hipMemcpy(d_delta_ll, h_delta_ll, sizeof(float) * (dense_output_M * 1), hipMemcpyHostToDevice);       
        hipMemcpy(d_output, &h_output[784*i], sizeof(float) * (output_N * output_N), hipMemcpyHostToDevice);
        hipMemcpy(d_train_image, &h_train_images[784*i], sizeof(float) * 784, hipMemcpyHostToDevice);
        hipMemcpy(d_dense_output, &h_dense_output[10*i], sizeof(float) * (dense_output_M * 1), hipMemcpyHostToDevice);
        hipMemcpy(d_train_label, one_hot_label, sizeof(int) * dense_output_M, hipMemcpyHostToDevice);

        
        dim3 gridsize(output_M);
        dim3 blocksize(output_M);
        convolutional_layer2D <<<gridsize, blocksize>>>(d_filter, d_train_image, d_output, d_bias_conv);
        hipFree(d_train_image);

        dim3 gridsize_sig(1);
        dim3 blocksize_sig(output_M*output_M);
        sigmoid_function<<<gridsize_sig, blocksize_sig>>>(d_output,d_output);

        //hipMemcpy(&h_output[784*i], d_output, sizeof(float) * (output_M * output_M), hipMemcpyDeviceToHost);
        //hipMemcpy(&h_output[784*i], d_output, sizeof(float) * (output_M * output_M), hipMemcpyDeviceToHost);
        
        dim3 gridsize_dense(1);
        dim3 blocksize_dense(dense_output_M);
        forward_propagation_fc<<<gridsize_dense, blocksize_dense>>>(d_output, d_weights, d_bias_dense, d_dense_output);

        dim3 gridsize_sig_dense(1);
        dim3 blocksize_sig_dense(dense_output_M * 1);
        sigmoid_function<<<gridsize_sig_dense, blocksize_sig_dense>>>(d_dense_output,d_dense_output);


        if (i == 0){
            //check_matrix(&h_train_images[784*i], input_M, input_M);
            //check_matrix(&h_output[784*i], output_M, output_M);
            //check_matrix(&h_dense_output[10*i], 1, dense_output_M);
            //check_matrix(h_weights,dense_output_M,output_M*output_M);
            //cout<<"Hello from 1"<<endl;
        }
        
        
        // Backprop for last layer
        dim3 gridsize_ll(1);
        dim3 blocksize_ll(dense_output_M * 1);
        backward_propagation_fc_lastlayer<<<gridsize_ll,blocksize_ll>>>(d_dense_output,d_train_label,d_delta_ll);
        
        
        // Backprop for previous layers
        dim3 gridsize_dense_bp(output_M*output_M);
        dim3 blocksize_dense_bp(dense_output_M * 1);
        backward_propagation_fc<<<gridsize_dense_bp,blocksize_dense_bp>>>(d_output,d_delta_ll,d_delta_curr,d_weights);
        hipMemcpy(h_delta_curr, d_delta_curr, sizeof(float) * (dense_output_M * (output_M * output_M)), hipMemcpyDeviceToHost);
        //hipMemcpy(&h_dense_output[10*i], d_dense_output, sizeof(float) * (dense_output_M * 1), hipMemcpyDeviceToHost);
        
        
        dim3 gridsize_wts_update(dense_output_M);
        dim3 blocksize_wts_update(output_M*output_M);
        weight_update<<<gridsize_wts_update,blocksize_wts_update>>>(d_delta_curr,d_weights);
        

        dim3 gridsize_dense_grad_input(1)
        dim3 blocksize_dense_grad_input(dense_output_M);

        input_grad<<<gridsize_dense_grad_input, blocksize_dense_grad_input>>>(d_dense_grad_input, d_dense_output);



        //hipMemcpy(h_weights, d_weights, sizeof(float) * (dense_output_M * (output_M * output_M)), hipMemcpyDeviceToHost);
        if (i == 0){
            //check_matrix(&h_train_images[784*i], input_M, input_M);
            //check_matrix(&h_output[784*i], output_M, output_M);
            //check_matrix(&h_dense_output[10*i], 1, dense_output_M);
            //check_matrix(h_delta_curr,dense_output_M,output_M*output_M);
            //cout<<"Hello from 2"<<endl;
            //check_matrix(h_weights,dense_output_M,output_M*output_M);
            //cout<<"weights from 2 yolooooooooo"<<endl;
        }
        
        hipFree(d_output);
        hipFree(d_train_label);
        hipFree(d_dense_output);
        hipFree(d_delta_ll);
        hipFree(d_delta_curr);
        free(h_delta_ll);
        free(one_hot_label);
        free(h_delta_curr);
    }
    hipFree(d_filter);
    hipFree(d_bias_conv);
    hipFree(d_weights);
    hipFree(d_bias_dense);
    // Free the memory.
    free(h_train_images);
    free(h_train_labels);
    free(h_test_images);
    free(h_test_labels);
    free(h_output);
    free(h_bias_conv);
    free(h_filter);
    free(h_weights);
    free(h_bias_dense);
    free(h_dense_output);
    return 0;

}