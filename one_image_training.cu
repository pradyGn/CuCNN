#include "Mnist_test.h"
#include "convolution.h"
#include "constants.h"
#include "dense.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

int main(){

    float* h_train_images = (float*)malloc(sizeof(float) * 60000 * 784);
    float* h_train_labels = (float*)malloc(sizeof(float) * 60000);
    float* h_test_images = (float*)malloc(sizeof(float) * 10000 * 784);
    float* h_test_labels = (float*)malloc(sizeof(float) * 10000);

    float  *h_output, *d_filter, *h_filter, *h_bias_conv, *d_bias_conv, *d_bias_dense, *h_bias_dense, *h_weights, *d_weights;

    float *h_dense_output;


    get_image_data(h_train_images, h_train_labels, h_test_images, h_test_labels);

    h_output = (float*)malloc(sizeof(float) * 60000 * (output_N * output_N));
    h_dense_output = (float*)malloc(sizeof(float) * 60000 * (dense_output_M * 1));

    // bias initization and allocation
    h_bias_conv = (float*)malloc(sizeof(float) * (filter_M * filter_M));
    initialize_filter(h_bias_conv, filter_M, filter_M);
    hipMalloc((void**)&d_bias_conv, sizeof(float) * (filter_M * filter_M));
    hipMemcpy(d_bias_conv, h_bias_conv, sizeof(float) * (filter_M * filter_M), hipMemcpyHostToDevice);
    //check_matrix(h_bias_conv, filter_M, filter_M);




    // filter initization and allocation
    h_filter = (float*)malloc(sizeof(float) * (filter_M * filter_M));
    initialize_filter(h_filter, filter_M, filter_M);
    hipMalloc((void**)&d_filter, sizeof(float) * (filter_M * filter_M));
    hipMemcpy(d_filter, h_filter, sizeof(float) * (filter_M * filter_M), hipMemcpyHostToDevice);
    //check_matrix(h_filter, filter_M, filter_M);

    // Initialize and allocate weights and bias for Dense layer
    h_weights = (float*)malloc(sizeof(float) * (dense_output_M * (output_M * output_M)));
    h_bias_dense = (float*)malloc(sizeof(float) * (dense_output_M));
    initialize_dense_weights_and_bias(h_weights, h_bias_dense);
    hipMalloc((void**)&d_weights, sizeof(float) * (dense_output_M * (output_M * output_M)));
    hipMalloc((void**)&d_bias_dense, sizeof(float) * dense_output_M);
    hipMemcpy(d_weights, h_weights, sizeof(float) * (dense_output_M * (output_M * output_M)), hipMemcpyHostToDevice);
    hipMemcpy(d_bias_dense, h_bias_dense, sizeof(float) * dense_output_M, hipMemcpyHostToDevice);
    check_matrix(h_bias_dense, 1, dense_output_M);
    check_matrix(h_weights, dense_output_M, (output_M * output_M));

    for (int i = 0; i < 2; i++){

        initialize_output(&h_output[784*i], output_N, output_N);
        
        initialize_dense_output(&h_dense_output[10*i]);
        
        float *d_train_image;
        float *d_dense_output, *d_output;

        hipMalloc((void**)&d_output, sizeof(float) * (output_N * output_N));
        hipMalloc((void**)&d_train_image, sizeof(float) * 784);
        hipMalloc((void**)&d_dense_output, sizeof(float) * dense_output_M);


        hipMemcpy(d_output, &h_output[784*i], sizeof(float) * (output_N * output_N), hipMemcpyHostToDevice);
        hipMemcpy(d_train_image, &h_train_images[784*i], sizeof(float) * 784, hipMemcpyHostToDevice);

        hipMemcpy(d_dense_output, &h_dense_output[10*i], sizeof(float) * (dense_output_M * 1), hipMemcpyHostToDevice);

        
        dim3 gridsize(output_M);
        dim3 blocksize(output_M);

        convolutional_layer2D <<<gridsize, blocksize>>>(d_filter, d_train_image, d_output, d_bias_conv);

        //hipMemcpy(&h_output[784*i], d_output, sizeof(float) * (output_M * output_M), hipMemcpyDeviceToHost);
        
        dim3 gridsize_dense(1);
        dim3 blocksize_dense(dense_output_M);

        forward_propagation_fc<<<gridsize_dense, blocksize_dense>>>(d_output, d_weights, d_bias_dense, d_dense_output);
        hipMemcpy(&h_output[784*i], d_output, sizeof(float) * (output_M * output_M), hipMemcpyDeviceToHost);
        hipMemcpy(&h_dense_output[10*i], d_dense_output, sizeof(float) * (dense_output_M * 1), hipMemcpyDeviceToHost);
        
        if (i == 1){
            //check_matrix(&h_train_images[784*i], input_M, input_M);
            check_matrix(&h_output[784*i], output_M, output_M);
            check_matrix(&h_dense_output[10*i], 1, dense_output_M);
        }

        hipFree(d_output);
        hipFree(d_dense_output);
        hipFree(d_train_image);

    }
    
    hipFree(d_filter);
    hipFree(d_bias_conv);
    hipFree(d_weights);
    hipFree(d_bias_dense);

    // Free the memory.
    free(h_train_images);
    free(h_train_labels);
    free(h_test_images);
    free(h_test_labels);
    free(h_output);
    free(h_bias_conv);
    free(h_filter);
    free(h_weights);
    free(h_bias_dense);
    free(h_output);

    return 0;

}