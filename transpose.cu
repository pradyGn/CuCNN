#include "hip/hip_runtime.h"
#include "convolution.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void transpose(float *d_odata, float *d_idata, int width, int height) {
  // Get the thread ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Calculate the global matrix indices
  int x = tid % width;
  int y = tid / width;

  // Calculate the 1D index of the element in the output matrix
  int index = y * width + x;

  // Write the element from the input matrix to the output matrix
  d_odata[index] = d_idata[tid];
}


int main() {

// Allocate memory on the device for the input and output matrices
float *idata, *odata;
float *d_idata, *d_odata;
int width = 4;
int height = 7;
idata = (float*)malloc(width * height * sizeof(float));
odata = (float*)malloc(7*4* sizeof(float));
hipMalloc((void **)&d_idata, 4 * 7 * sizeof(float));
hipMalloc((void **)&d_odata, 4 * 7 * sizeof(float));

initialize_filter(idata, 4,7);
check_matrix(idata,4,7);
// Copy the input matrix to the device
hipMemcpy(d_idata, idata, width * height * sizeof(float), hipMemcpyHostToDevice);

// Launch the kernel
transpose<<<4, 7>>>(d_odata, d_idata, width, height);

// Copy the output matrix back to the host
hipMemcpy(odata, d_odata, width * height * sizeof(float), hipMemcpyDeviceToHost);
check_matrix(odata,7,4);
// Free the device memory
hipFree(d_idata);
hipFree(d_odata);

free(idata);
free(odata);

return 0;
}