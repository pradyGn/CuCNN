#include "hip/hip_runtime.h"
#include "constants.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void transpose(float *matrix_t, float *matrix){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.x;

    matrix_t[i] = matrix[j + threadIdx.x * blockDim.x];
}


void initialize_output(float *matrix, int matrix_M, int matrix_N){
    for (int i = 0; i < matrix_M; i++){
        for (int j = 0; j < matrix_N; j++){
            matrix[(i*matrix_N) + j] = i+j;
        }
    }
}


void check_matrix(float *matrix, int matrix_M, int matrix_N){
    for (int i=0; i<matrix_M; i++){
        for (int j=0; j<matrix_N; j++)
        {
                printf("%.2f", matrix[(i*matrix_M)+j]);
                printf(" ");
        }
        printf("\n");
    }
    printf("\n");
}



int main(){

    float *d_output, *d_inpu;
    float *output = (float*)malloc(sizeof(float) * 4 * 7);
    float *input = (float*)malloc(sizeof(float) * 4 * 7);
    initialize_output(input, 4, 7);

    check_matrix(input, 4, 7);

    hipMalloc((void**)&d_input, sizeof(float) * (4 * 7));
    hipMalloc((void**)&d_output, sizeof(float) * (7 * 4));
    
    hipMemcpy(d_input, input, sizeof(float) * (7 * 4), hipMemcpyHostToDevice);



    dim3 griddim(4);
    dim3 blockdim(7);

    transpose<<<griddim, blockdim>>>(d_output, d_input);


    hipMemcpy(output, d_output, sizeof(float) * (7 * 4), hipMemcpyDeviceToHost);

    check_matrix(output, 7, 4);

    hipFree(d_output);
    hipFree(d_input);

    free(output);
    free(input);



    return 0;


}